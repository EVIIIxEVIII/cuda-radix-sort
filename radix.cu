#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCK_SIZE 32
#define N 134217728
#define RADIX 2

#define CHECK_CUDA(call)                                                      \
    {                                                                         \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,  \
                    hipGetErrorString(err));                                  \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    }

double getTimeMicroseconds() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec * 1e6 + ts.tv_nsec / 1e3;
}

void init_array(int* arr, int n) {
    for (int i = 0; i < n; i++) {
        arr[i] = rand();
    }
}

__global__ void compute_global_count(int* arr, int* global_count, int n, int iter) {
    __shared__ int local_count_map[4];

    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (tid < 4) local_count_map[tid] = 0;

    __syncthreads();

    int digit = 0;
    int active = (gid < n);
    if (active) {
        digit = (arr[gid] >> (2 * iter)) & 3;
    }

    int mask_0 = __ballot_sync(0xFFFFFFFF, digit == 0);
    int mask_1 = __ballot_sync(0xFFFFFFFF, digit == 1);
    int mask_2 = __ballot_sync(0xFFFFFFFF, digit == 2);
    int mask_3 = __ballot_sync(0xFFFFFFFF, digit == 3);

    int count_0 = __popc(mask_0);
    int count_1 = __popc(mask_1);
    int count_2 = __popc(mask_2);
    int count_3 = __popc(mask_3);

    if (tid == 0) {
        local_count_map[0] += count_0;
        local_count_map[1] += count_1;
        local_count_map[2] += count_2;
        local_count_map[3] += count_3;
    }

    __syncthreads();

    if (tid < 4) {
        global_count[gridDim.x * tid + blockIdx.x] = local_count_map[tid];
    }
}
__global__ void in_lane_scan(int* arr, int* in_lane_scans, int* sums, int n) {
    __shared__ int local_offset[BLOCK_SIZE];
    __shared__ int temp[BLOCK_SIZE];
    __shared__ int sum;

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid == 0) sum = 0;
    if (gid < n) {
        temp[tid] = (tid == 0) ? 0 : arr[gid - 1];
    }

    if (gid < n) {
        atomicAdd(&sum, arr[gid]);
    }

    __syncthreads();

    if (tid < BLOCK_SIZE) {
        for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
            int val = (tid >= offset) ? temp[tid - offset] : 0;
            __syncthreads();
            temp[tid] += val;
            __syncthreads();
        }

        local_offset[tid] = temp[tid];
    }

    __syncthreads();

    if (gid < n) {
        in_lane_scans[gid] = local_offset[tid];
    }

    if (tid == 0) {
        sums[bid] = sum;
    }
}

__global__ void parallel_scan_sums(int* sums, int* temp, int sums_num) {


}

__global__ void in_lane_propagation(int* in_lane_scans, int* sums, int* res, int n) {
    const int bid = blockIdx.x;
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < n) {
        res[gid] = in_lane_scans[gid] + sums[bid];
    }
}


__global__ void radix_sort(int* arr, int* res, int* global_offset, int n, int iter) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < n) {
        int digit = (arr[gid] >> (2 * iter)) & 3;
        int final_idx = atomicAdd(&global_offset[digit * gridDim.x + blockIdx.x], 1);
        res[final_idx] = arr[gid];
    }
}

int main() {
    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int const maps_size = 4 * gridDim.x;

    dim3 blockDimILS(BLOCK_SIZE);
    dim3 gridDimILS((maps_size + BLOCK_SIZE - 1) / BLOCK_SIZE);

    printf("Threads num (radix): %d \n", blockDim.x * gridDim.x);
    printf("Blocks num  (radix): %d \n", gridDim.x);

    printf("Threads num (ils): %d \n", blockDimILS.x * gridDimILS.x);
    printf("Blocks num  (ils): %d \n", gridDimILS.x);

    int* h_arr = (int*)malloc(N * sizeof(int));
    int* h_res = (int*)malloc(N * sizeof(int));
    int* h_global_count = (int*)malloc(maps_size * sizeof(int));
    int* h_global_offset = (int*)malloc(maps_size * sizeof(int));

    int* h_sums = (int*)malloc(gridDim.x * sizeof(int));
    int* h_sums_offsets = (int*)malloc(gridDim.x * sizeof(int));

    init_array(h_arr, N);

    int* d_arr;
    int* d_res;
    int* d_global_count;
    int* d_global_offset;
    int* d_sums;
    int* d_in_lane_scans;

    hipMalloc(&d_arr, N * sizeof(int));
    hipMalloc(&d_res, N * sizeof(int));
    hipMalloc(&d_global_count,  maps_size * sizeof(int));
    hipMalloc(&d_global_offset, maps_size * sizeof(int));
    hipMalloc(&d_in_lane_scans, maps_size * sizeof(int));
    hipMalloc(&d_sums, gridDim.x * sizeof(int));

    hipMemset(d_global_offset, 0, maps_size * sizeof(int));
    hipMemset(d_global_count,  0, maps_size * sizeof(int));
    hipMemset(d_in_lane_scans, 0, maps_size * sizeof(int));
    hipMemset(d_sums,          0, gridDim.x * sizeof(int));

    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    double totalGc = 0;
    double totalIls = 0;
    double totalIlp = 0;
    double totalCpu = 0;
    double totalRadix = 0;

    hipFree(0);
    double start = getTimeMicroseconds();
    for (int i = 0; i < 16; i++) {
        double gcStart = getTimeMicroseconds();
        compute_global_count<<<gridDim, blockDim>>>(d_arr, d_global_count, N, i);
        CHECK_CUDA(hipDeviceSynchronize());
        double gcEnd = getTimeMicroseconds();

        double ilsStart = getTimeMicroseconds();
        in_lane_scan<<<gridDimILS, blockDimILS>>>(d_global_count, d_in_lane_scans, d_sums, maps_size);
        CHECK_CUDA(hipDeviceSynchronize());
        double ilsEnd = getTimeMicroseconds();

        double startCpu = getTimeMicroseconds();
        hipMemcpy(h_sums, d_sums, gridDim.x * sizeof(int), hipMemcpyDeviceToHost);
        h_sums_offsets[0] = 0;
        for (int j = 1; j < gridDim.x; j++) {
            h_sums_offsets[j] = h_sums_offsets[j - 1] + h_sums[j - 1];
        }
        hipMemcpy(d_sums, h_sums_offsets, gridDim.x * sizeof(int), hipMemcpyHostToDevice);
        double endCpu = getTimeMicroseconds();

        double ilpStart = getTimeMicroseconds();
        in_lane_propagation<<<gridDimILS, blockDimILS>>>(d_in_lane_scans, d_sums, d_global_offset, maps_size);
        CHECK_CUDA(hipDeviceSynchronize());
        double ilpEnd = getTimeMicroseconds();

        double radixStart = getTimeMicroseconds();
        radix_sort<<<gridDim, blockDim>>>(d_arr, d_res, d_global_offset, N, i);
        CHECK_CUDA(hipDeviceSynchronize());
        double radixEnd = getTimeMicroseconds();

        hipMemcpy(d_arr, d_res, N * sizeof(int), hipMemcpyDeviceToDevice);

        totalGc += (gcEnd - gcStart);
        totalIls += (ilsEnd - ilsStart);
        totalIlp += (ilpEnd - ilpStart);
        totalCpu += (endCpu - startCpu);
        totalRadix += (radixEnd - radixStart);
    }
    double end = getTimeMicroseconds();

    printf("\n\nTime taken (total): %lf microseconds\n\n", end - start);

    printf("Time taken (gc)   : %lf microseconds\n", totalGc);
    printf("Time taken (ils)  : %lf microseconds\n", totalIls);
    printf("Time taken (ilp)  : %lf microseconds\n", totalIlp);
    printf("Time taken (cpu)  : %lf microseconds\n", totalCpu);
    printf("Time taken (radix): %lf microseconds\n", totalRadix);

    hipMemcpy(h_res, d_res, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_global_count, d_global_count, maps_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_global_offset, d_global_offset, maps_size * sizeof(int), hipMemcpyDeviceToHost);

    //printf("\nSorted array: \n");
    //for (int i = 0; i < N; i++) {
    //    printf("%d \n", h_res[i]);
    //}

    printf("\n\n");
    for (int i = 0; i < N-1; i++) {
        if(h_res[i] > h_res[i+1]) {
            printf("FAILED TO SORT THE ARRAY %d < %d", h_res[i], h_res[i+1]);
            printf("\n\n");
            return 0;
        }
    }

    printf("\n\nTEST PASSED! SORTING PERFORMED CORRECTLY!\n\n");

    free(h_arr);
    free(h_res);

    hipFree(d_arr);
    hipFree(d_res);
    hipFree(d_global_offset);
    hipFree(d_global_count);
}

